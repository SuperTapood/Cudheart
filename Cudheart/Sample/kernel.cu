#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime>

using namespace std;

std::chrono::duration<double> addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b, int n)
{
	int i = threadIdx.x + blockIdx.x * blockDim.x;

	int v;
	for (int j = 0; j < 500; j++) {
		v = logf(a[i]) + logf(b[i]);
		v = sqrtf(v);
		int s = 50;
		for (int i = 3; i < s; i += i)
		{
			v += 1;
		}
	}
	if (i < n) {
		c[i] = a[i] + b[i];
	}
}

int func()
{
	unsigned long long arraySize = 50000;
	int* a = new int[arraySize];
	int* b = new int[arraySize];
	int* ca = new int[arraySize];
	int* cb = new int[arraySize];

	for (int i = 0; i < arraySize; i++) {
		a[i] = (i + 1);
		b[i] = (i + 1) * 10;
	}
	printf("cuda start\n");

	// Add vectors in parallel.
	std::chrono::duration<double> cuda = addWithCuda(ca, a, b, arraySize);

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%f,%f,%f,%f,%f}\n",
		//c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	hipError_t cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf("cpp start\n");
	auto start = std::chrono::system_clock::now();
	for (int i = 0; i < arraySize; i++) {
		int v;
		for (int j = 0; j < 500; j++) {
			v = logf(a[i]) + logf(b[i]);
			v = sqrtf(v);
			int s = 50;
			for (int i = 3; i < s; i += i)
			{
				v += 1;
			}
		}
		cb[i] = a[i] + b[i];

		cout << "\ncomputing i " << i;
	}

	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);

	std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

	for (int i = 0; i < arraySize; i++) {
		printf("comparing i %d\n", i);
		if (ca[i] != cb[i]) {
			printf("%d: %d + %d = ", i, a[i], b[i]);
			printf("%d == %d\n", ca[i], cb[i]);
			printf("oof\n");
			break;
		}
	}

	std::cout << "diff is " << elapsed_seconds.count() / cuda.count() << "x" << std::endl;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
std::chrono::duration<double> addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	hipDeviceProp_t stDeviceProp;
	hipGetDeviceProperties(&stDeviceProp, 0);
	if (!stDeviceProp.unifiedAddressing) {
		fprintf(stderr, "fail");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	printf("Using device %d: %s\n", 0, stDeviceProp.name);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	int threads = 1024;

	auto start = std::chrono::system_clock::now();

	int N = floor(size / threads) + 1;

	// Launch a kernel on the GPU with one thread for each element.
	addKernel << <N, threads >> > (dev_c, dev_a, dev_b, size);

	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);

	std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %f after launching addKernel!\n", cudaStatus);
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		hipFree(dev_c);
		hipFree(dev_a);
		hipFree(dev_b);
		std::exit(69);
	}

	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return elapsed_seconds;
}