#include "hip/hip_runtime.h"
﻿#include "kernel.cuh"

#include "hip/hip_runtime.h"
#include ""
#include "hip/hip_runtime.h"

#include <stdio.h>
#include <iostream>
#include <chrono>
#include <ctime> 

using namespace std;

hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size);

__global__ void addKernel(int* c, const int* a, const int* b)
{
	int i = blockIdx.x;
	/*
	c[i] = logf(a[i]) + logf(b[i]);
	c[i] = sqrtf(c[i]);
	int s = 50;
	for (int i = 3; i < s; i+=i)
	{
		c[i] += 1;
	}
	*/
	c[i] = a[i] + b[i];
	// printf("%d: %d + %d = %d\n", i, a[i], b[i], c[i]);
}

int func()
{
	const int arraySize = 70000;
	int a[arraySize];
	int b[arraySize];
	int c[arraySize];
	
	for (int i = 0; i < arraySize; i++) {
		a[i] = (i + 1);
		b[i] = (i + 1) * 10;
	}
	printf("cuda start\n");	

	// Add vectors in parallel.
	hipError_t cudaStatus = addWithCuda(c, a, b, arraySize);
	
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addWithCuda failed!");
		return 1;
	}

	//printf("{1,2,3,4,5} + {10,20,30,40,50} = {%d,%d,%d,%d,%d}\n",
		//c[0], c[1], c[2], c[3], c[4]);

	// hipDeviceReset must be called before exiting in order for profiling and
	// tracing tools such as Nsight and Visual Profiler to show complete traces.
	cudaStatus = hipDeviceReset();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceReset failed!");
		return 1;
	}

	printf("cpp start\n");
	auto start = std::chrono::system_clock::now();
	for (int i = 0; i < arraySize; i++) {
		c[i] = a[i] + b[i];
	}
	
	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);

	std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

	return 0;
}

// Helper function for using CUDA to add vectors in parallel.
hipError_t addWithCuda(int* c, const int* a, const int* b, unsigned int size)
{
	int* dev_a = 0;
	int* dev_b = 0;
	int* dev_c = 0;
	hipError_t cudaStatus;

	// Choose which GPU to run on, change this on a multi-GPU system.
	cudaStatus = hipSetDevice(0);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipSetDevice failed!  Do you have a CUDA-capable GPU installed?");
		goto Error;
	}
	
	hipDeviceProp_t stDeviceProp;
	hipGetDeviceProperties(&stDeviceProp, 0);
	if (!stDeviceProp.unifiedAddressing) {
		fprintf(stderr, "fail");
		goto Error;
	}

	printf("Using device %d: %s\n", 0, stDeviceProp.name);

	// Allocate GPU buffers for three vectors (two input, one output)    .
	cudaStatus = hipMalloc((void**)&dev_c, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_a, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	cudaStatus = hipMalloc((void**)&dev_b, size * sizeof(int));
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMalloc failed!");
		goto Error;
	}

	// Copy input vectors from host memory to GPU buffers.
	cudaStatus = hipMemcpy(dev_a, a, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	cudaStatus = hipMemcpy(dev_b, b, size * sizeof(int), hipMemcpyHostToDevice);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

	auto start = std::chrono::system_clock::now();
	
	// Launch a kernel on the GPU with one thread for each element.
	addKernel <<<size, 1>>> (dev_c, dev_a, dev_b);

	auto end = std::chrono::system_clock::now();
	std::chrono::duration<double> elapsed_seconds = end - start;
	std::time_t end_time = std::chrono::system_clock::to_time_t(end);

	std::cout << "elapsed time: " << elapsed_seconds.count() << "s" << std::endl;

	// Check for any errors launching the kernel
	cudaStatus = hipGetLastError();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "addKernel launch failed: %s\n", hipGetErrorString(cudaStatus));
		goto Error;
	}

	// hipDeviceSynchronize waits for the kernel to finish, and returns
	// any errors encountered during the launch.
	cudaStatus = hipDeviceSynchronize();
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipDeviceSynchronize returned error code %d after launching addKernel!\n", cudaStatus);
		goto Error;
	}

	// Copy output vector from GPU buffer to host memory.
	cudaStatus = hipMemcpy(c, dev_c, size * sizeof(int), hipMemcpyDeviceToHost);
	if (cudaStatus != hipSuccess) {
		fprintf(stderr, "hipMemcpy failed!");
		goto Error;
	}

Error:
	hipFree(dev_c);
	hipFree(dev_a);
	hipFree(dev_b);

	return cudaStatus;
}